#include "hip/hip_runtime.h"
#include ""
#include <iostream>

using namespace std;

void GetCudaImfo() {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  int dev;
  for (dev = 0; dev < deviceCount; dev++) {
    int driver_version(0), runtime_version(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (dev == 0)
      if (deviceProp.minor = 9999 && deviceProp.major == 9999)
        cout << endl;
    cout << "使用GPU device " << dev << ": " << deviceProp.name << endl;
    hipDriverGetVersion(&driver_version);
    cout << "CUDA驱动版本:" << driver_version / 1000 << "."
         << (driver_version % 1000) / 10 << endl;
    hipRuntimeGetVersion(&runtime_version);
    cout << "CUDA运行时版本:" << runtime_version / 1000 << "."
         << (runtime_version % 1000) / 10 << endl;
    cout << "设备计算能力:" << deviceProp.major << "." << deviceProp.minor
         << endl;
    cout << "显卡时钟频率:" << deviceProp.clockRate * 1e-6f << " GHz" << endl;
    cout << "内存时钟频率:" << deviceProp.memoryClockRate * 1e-3f << " MHz"
         << endl;
    cout << "内存总线带宽:" << deviceProp.memoryBusWidth << " bit" << endl;
    cout << "总显存大小:" << deviceProp.totalGlobalMem / (1024.0 * 1024.0)
         << " MB" << endl;
    cout << "总常量内存大小:" << deviceProp.totalConstMem / 1024.0 << " KB"
         << endl;
    cout << "SM数量:" << deviceProp.multiProcessorCount << endl;
    cout << "每个SM最大线程数:" << deviceProp.maxThreadsPerMultiProcessor
         << endl;
    cout << "每个线程块(block)共享内存大小:"
         << deviceProp.sharedMemPerBlock / 1024.0 << " KB" << endl;
    cout << "每个线程块(block)的最大线程数:" << deviceProp.maxThreadsPerBlock
         << endl;
    cout << "每个线程块(block)的最大可用寄存器数:" << deviceProp.regsPerBlock
         << endl;
    cout << "线程束(wrap)尺寸:" << deviceProp.warpSize << endl;
    cout << "每个线程块(block)各个维度最大尺寸:" << deviceProp.maxThreadsDim[0]
         << " x " << deviceProp.maxThreadsDim[1] << " x "
         << deviceProp.maxThreadsDim[2] << endl;
    cout << "每个线程格(grid)各个维度最大尺寸:" << deviceProp.maxGridSize[0]
         << " x " << deviceProp.maxGridSize[1] << " x "
         << deviceProp.maxGridSize[2] << endl;
    cout << "最大存储间距:" << deviceProp.memPitch / (1024.0 * 1024.0) << " MB"
         << endl;
  }
}

int main() {
  GetCudaImfo();

  return 0;
}